#include "hip/hip_runtime.h"
#include "misc_kernels_impl.cuh"
#include "misc_kernels.h"
#include "dispatch_utils.h"

namespace nunchaku::kernels {

Tensor add(Tensor a, Tensor b) {
    assert(a.shape.dataExtent == b.shape.dataExtent);
    assert(a.dtype() == b.dtype());
    assert(a.is_contiguous());
    assert(b.is_contiguous());

    int threadsPerBlock = 1024;
    int blocksPerGrid = (a.numel() + threadsPerBlock - 1) / threadsPerBlock;

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty_like(a);

    dispatch(out.scalar_type(), [&]<typename scalar_t>() {
        add_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            a.data_ptr<scalar_t>(), b.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), out.numel());
    });

    return out;
}

void mul_add(Tensor x, Tensor scale, Tensor bias) {
    // assert(scale.shape.data == bias.shape.data);
    // FIXME FIXME
    assert(x.numel() % scale.numel() == 0);
    assert(x.numel() % bias.numel() == 0);
    assert(x.dtype() == scale.dtype());
    assert(x.dtype() == bias.dtype());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert(!scale.valid() || (uintptr_t)scale.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert((uintptr_t)bias.data_ptr() % (x.scalar_size() * unroll) == 0);

    assert(x.numel() % unroll == 0);
    assert(!scale.valid() || scale.numel() % unroll == 0);
    assert(bias.numel() % unroll == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        if (scale.valid()) {
            mul_add_kernel<scalar_t, unroll, false><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
                x.data_ptr<scalar_t>(), scale.data_ptr<scalar_t>(), bias.data_ptr<scalar_t>(), 0, x.numel(), scale.numel(), bias.numel(), 0, 0, 0);
        } else {
            mul_add_kernel<scalar_t, unroll, true><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
                x.data_ptr<scalar_t>(), nullptr, bias.data_ptr<scalar_t>(), 0, x.numel(), 1, bias.numel(), 0, 0, 0);
        }
    });
}

void mul_add_batch(Tensor x, Tensor scale, bool batch_scale, double scale_shift, Tensor bias, bool batch_bias) {

    const int batch_size = x.shape[0];
    assert(!batch_scale || scale.shape[0] == batch_size);
    assert(!batch_bias || bias.shape[0] == batch_size);

    const int numel = x.numel() / batch_size;
    const int numel_scale = scale.valid() ? (scale.numel() / (batch_scale ? batch_size : 1)) : 1;
    const int numel_bias  = bias.numel() / (batch_bias ? batch_size : 1);

    assert(numel % numel_scale == 0);
    assert(numel % numel_bias == 0);
    assert(!scale.valid() || x.dtype() == scale.dtype());
    assert(x.dtype() == bias.dtype());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert(!scale.valid() || (uintptr_t)scale.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert((uintptr_t)bias.data_ptr() % (x.scalar_size() * unroll) == 0);

    assert(numel % unroll == 0);
    assert(!scale.valid() || numel_scale % unroll == 0);
    assert(numel_bias % unroll == 0);

    int threadsPerBlock = 1024;
    dim3 grid(ceilDiv(numel, threadsPerBlock * unroll), batch_size);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        if (scale.valid()) {
            mul_add_kernel<scalar_t, unroll, false><<<grid, threadsPerBlock, 0, stream>>>(
                x.data_ptr<scalar_t>(), scale.data_ptr<scalar_t>(), bias.data_ptr<scalar_t>(), 
                (scalar_t)scale_shift,
                numel, numel_scale, numel_bias, 
                x.stride(0), batch_scale ? scale.stride(0) : 0, batch_bias ? bias.stride(0) : 0);
        } else {
            mul_add_kernel<scalar_t, unroll, true><<<grid, threadsPerBlock, 0, stream>>>(
                x.data_ptr<scalar_t>(), nullptr, bias.data_ptr<scalar_t>(), 
                (scalar_t)scale_shift,
                numel, 1, numel_bias, 
                x.stride(0), 0, batch_bias ? bias.stride(0) : 0);
        }
    });
}

Tensor embedding(Tensor input_id, Tensor lookup) {
    assert(input_id.dtype() == Tensor::INT32);
    assert(lookup.ndims() == 2);

    auto shapeOut = input_id.shape;
    shapeOut.dataExtent.push_back(lookup.shape[-1]);

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty(shapeOut, lookup.scalar_type(), input_id.device());

    dispatch(out.scalar_type(), [&]<typename scalar_t>() {
        EmbeddingKernel<<<input_id.numel(), std::min(lookup.shape[-1], 1024), 0, stream>>>(
            input_id.data_ptr<int32_t>(), out.data_ptr<scalar_t>(), lookup.data_ptr<scalar_t>(), lookup.shape[-1]);
    });

    return out;
}

Tensor argmax_sample(Tensor logits) {
    assert(logits.ndims() == 2);

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty({logits.shape[0]}, Tensor::INT32, logits.device());

    dispatch(logits.scalar_type(), [&]<typename scalar_t>() {
        argmax_sample_kernel<<<logits.shape[0], std::min(logits.shape[1], 1024), 0, stream>>>(
            logits.data_ptr<scalar_t>(), out.data_ptr<int32_t>(), logits.shape[1]
        );
    });

    return out;
}

void splitqkv(Tensor qkv, Tensor q, Tensor k, Tensor v) {
    // FIXME FIXME
    // assert(qkv.shape[0] == q.shape[0]);
    // assert(qkv.shape[0] == k.shape[0]);
    // assert(qkv.shape[0] == v.shape[0]);

    auto stream = getCurrentCUDAStream();

    int dim_q = q.shape[-1] * q.shape[-2];
    int dim_k = k.shape[-1] * k.shape[-2];
    int dim_v = v.shape[-1] * v.shape[-2];

    assert(dim_k == dim_v);
    assert(dim_q + dim_k + dim_v == qkv.shape[-1]);
    
    int num_tokens = qkv.numel() / qkv.shape[-1];

    dispatch(qkv.scalar_type(), [&]<typename scalar_t>() {
        splitqkv_kernel<<<num_tokens, std::min(qkv.shape[-1], 1024), 0, stream>>>(
            qkv.data_ptr<scalar_t>(),
            q.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            v.data_ptr<scalar_t>(),
            dim_q,
            dim_k
        );
    });

}

template<size_t N>
std::array<Tensor, N> split_mod(Tensor input) {
    assert(input.shape[-1] % N == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (input.numel() + threadsPerBlock - 1) / threadsPerBlock;

    auto stream = getCurrentCUDAStream();

    auto shapeOut = input.shape;
    shapeOut[-1] /= N;

    std::array<Tensor, N> out;
    for (int k = 0; k < N; k++) {
        out[k] = Tensor::empty(shapeOut, input.scalar_type(), input.device());
    }

    dispatch(input.scalar_type(), [&]<typename scalar_t>() {
        std::array<scalar_t *, N> outPtr;
        for (int k = 0; k < N; k++) {
            outPtr[k] = out[k].template data_ptr<scalar_t>();
        }
        split_mod_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            input.data_ptr<scalar_t>(),
            outPtr, input.numel());
    });

    return out;
}

Tensor quant_static(Tensor x, float scale) {
    Tensor out = Tensor::empty(x.shape, Tensor::INT8, x.device());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        quant_kernel_static<scalar_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            x.data_ptr<scalar_t>(), out.data_ptr<int8_t>(), (scalar_t)scale, x.numel());
    });

    return out;
}

Tensor quant_static_fuse_gelu(Tensor x, float scale) {
    Tensor out = Tensor::empty(x.shape, Tensor::INT8, x.device());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);


    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        quant_kernel_static_fuse_gelu<scalar_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            x.data_ptr<scalar_t>(), out.data_ptr<int8_t>(), (scalar_t)scale, x.numel());
    });

    return out;
}

void cast(Tensor input, Tensor output) {
    assert(input.is_contiguous());
    assert(output.is_contiguous());
    assert(input.shape.dataExtent == output.shape.dataExtent);

    if (input.data_ptr() == output.data_ptr()) {
        assert(input.scalar_size() == output.scalar_size());
    }

    auto stream = getCurrentCUDAStream();

    dispatch(input.scalar_type(), [&]<typename input_t>() {
        dispatch(output.scalar_type(), [&]<typename output_t>() {
            constexpr int unroll = 16 / std::max(sizeof(input_t), sizeof(output_t));

            int threadsPerBlock = 1024;
            int blocksPerGrid = (int)ceilDiv<int64_t>(input.numel(), threadsPerBlock * unroll);

            cast_kernel<input_t, output_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
                input.data_ptr<input_t>(), output.data_ptr<output_t>(), input.numel());

            checkCUDA(hipGetLastError());
        });
    });
}

Tensor topk(Tensor x, int k) {
    constexpr int MAXK = 64 + 4;

    const int N = x.shape[-1];
    const int batch = x.numel() / N;

    assert(k <= N);
    assert(k <= MAXK);

    auto outShape = TensorShape(x.shape.dataExtent);
    outShape[-1] = k;
    outShape.dataStride.clear();


    Tensor out = Tensor::empty(outShape, Tensor::INT32, x.device());

    auto stream = getCurrentCUDAStream();

    dispatchVal(k, std::make_integer_sequence<int, MAXK + 1>(), [&]<int K>() {
        if constexpr (K == 0) {
            assert(false);
            return;
        }
        if constexpr (K > 0) {
            dispatch(x.scalar_type(), [&]<typename scalar_t>() {
                topk_kernel<scalar_t, K><<<ceilDiv(batch, 32), 32, 0, stream>>>(
                    x.data_ptr<scalar_t>(),
                    out.data_ptr<int>(),
                    N, x.stride(-2), batch
                );
                checkCUDA(hipGetLastError());
            });
        }
    });

    return out;
}

template std::array<Tensor, 2> split_mod<2>(Tensor input);
template std::array<Tensor, 3> split_mod<3>(Tensor input);
template std::array<Tensor, 4> split_mod<4>(Tensor input);
template std::array<Tensor, 5> split_mod<5>(Tensor input);
template std::array<Tensor, 6> split_mod<6>(Tensor input);

};  // namespace nunchaku::kernels